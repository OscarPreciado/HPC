#include <stdio.h>
#include <iostream>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define TILE_WIDTH 32


using namespace std;
// Multiplicacion con shared mem
__global__ void matrixMulKernelTiled(float *d_M, float *d_N, float *d_P, int width1, int height1, int width2) {
  __shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
  __shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int row = by * TILE_WIDTH + ty;
  int col = bx * TILE_WIDTH + tx;

  float Pvalue = 0;

  for (int p = 0; p < width1 / TILE_WIDTH; p++) {
	 //Nos ubicamos en el elemento de la matriz 1 que deseamos multiplicar
    if (row < height1 and (p * TILE_WIDTH + tx) < width1) {
      ds_M[ty][tx] = d_M[row * width1 + p * TILE_WIDTH + tx];
    } else {
    //si esta fuera del rango llenamos con cero
      ds_M[ty][tx] = 0.0;
    }
         //Nos ubicamos en el elemento de la matriz 2 que deseamos multiplicar
    if ((p * TILE_WIDTH + ty) < width1 and col < width1) {
      ds_N[ty][tx] = d_N[(p * TILE_WIDTH + ty) * width2 + col];
    } else {
	//si esta fuera del rango llenamos con cero
      ds_N[ty][tx] = 0.0;
    }
    __syncthreads();
       //Se hace la multiplicacion utilizando shared mem
    if (row < height1 and col < width2)
      for (int k = 0; k < TILE_WIDTH; k++) {
        Pvalue += ds_M[ty][k] * ds_N[k][tx];
      }
    __syncthreads();
  }
  //Se guardan los resultados.
  if (row < height1 and col < width2)
    d_P[row * width2 + col] = Pvalue;
}

//Multiplicacón en GPU: 
void MatrixMulCPU(float *M, float *N, float *P, int width1, int height1, int width2) {
  //Aqui se guarda el resultado de la multiplicacion
  int sum = 0;
  for (int i = 0; i < height1; i++) {
    for (int j = 0; j < width2; j++) {
      sum = 0;
      for (int k = 0; k < width1; k++)
        //Se hace el productto y se guarda en la variable
        sum += M[i * width1 + k] * N[k * width2 + j];
      //Se colocan los valores en la matriz resultado  
      P[i * width2 + j] = sum; 
    }
  }
}


//Inicializa las matrices a multiplicar. 
int initValues(float *data, int width, int heigth){
    for(int i = 0; i < width*heigth; i++)
        data[i] = 1.0; 
    return 0;
}


int main()
{
  
    clock_t start, end;
    float *h_M, *h_N, *h_P,*h_P_d; //Matrices del host
    float *d_M, *d_N,*d_P; // Matrices del device

    //Aqui introducimos los tamaños de las matrices 1 y 2 (heigth y width)
  
    int heigth1 = 10;
    int width1 = 10; 
    int heigth2 = 10;
    int width2 = 15; 
  
    hipError_t error = hipSuccess; 
    
    int size1 = width1 * heigth1 * sizeof(float); //Dimension de la matriz 1
    int size2 = width2 * heigth2 * sizeof(float); //Dimension de la matriz 2
    int size3 = width2 * heigth1 * sizeof(float); //Dimension de la matriz resultado

	//Reservamos memoria para las matrices del host
    h_M = (float*)malloc(size1);
    h_N = (float*)malloc(size2);
    h_P = (float*)malloc(size3);
    h_P_d = (float*)malloc(size3);

    if(h_P_d == NULL)
        return 0;
	//Inicializamos las matrices 
    initValues(h_M, width1, heigth1);
    initValues(h_N, width2, heigth2);
  
	//Procedimiento en GPU:
  
   //Reservamos espacio en el device para una matriz  de dimensión size1
   error = hipMalloc((void**)&d_M,size1);
    if(error != hipSuccess){
        printf("Error reservando memoria para d_M");
        exit(0);
    }
	//Reservamos espacio en el device para una matriz  de dimensión size2
    error = hipMalloc((void**)&d_N,size2);
    if(error != hipSuccess){
        printf("Error reservando memoria para d_N");
        exit(0);
    }
	//Reservamos espacio en el device para la matriz resultante de size3
    error = hipMalloc((void**)&d_P,size3);
    if(error != hipSuccess){
        printf("Error reservando memoria para d_P");
        exit(0);
    }
	
	//Copiamos los datos de las matrices del host al device con las mismas dimensiones.
	error = hipMemcpy(d_M, h_M, size1, hipMemcpyHostToDevice);
	if(error != hipSuccess){
        printf("Error copiando datos a d_M");
        exit(0);
    }
    
    error = hipMemcpy(d_N, h_N, size2, hipMemcpyHostToDevice);
    if(error != hipSuccess){
        printf("Error copiando datos a d_N");
        exit(0);
    }
    
    int blockSize = 1;
    dim3 dimBlock(blockSize,blockSize,1); 
    dim3 dimGrid(ceil(width2 / float(blockSize)), ceil(heigth1 / float(blockSize)), 1); 
  
  
    // CICLO DE TIEMPOS
    for(int x=1; x<=5;x++)
    {
    printf ("Ciclo numero %d\n",x);  

    //multiplicación con CPU
    start = clock();
    MatrixMulCPU(h_M, h_N, h_P, width1, heigth1, width2); //Invocamos la multiplicacion secuencial en CPU.
    end = clock();
    double cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Tiempo en CPU: %.10f\n", cpu_time_used);
    //Fin
    
    //Multiplicacion con GPU
    start = clock();
  	matrixMulKernelTiled<<<dimGrid, dimBlock>>>(d_M, d_N, d_P, width1, heigth1, width2);// Invocamos la multiplicacion con Tiles.
    hipMemcpy(h_P_d,d_P,size3,hipMemcpyDeviceToHost); //Copiamos el resultado de la matriz  del device al host.
    end = clock();
    double gpu_time_used = double(end - start) / CLOCKS_PER_SEC;
    printf("Tiempo en GPU: %.10f\n",gpu_time_used);
    //FIN
    }

  	  	
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);
    

    return 0;
}
